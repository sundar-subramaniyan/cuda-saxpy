#include "hip/hip_runtime.h"
/**
 * @file saxpy.cu
 * @brief This file contains a basic CUDA programming example
 *	  Reference: https://developer.nvidia.com/blog/easy-introduction-cuda-c-and-c
 *
 * @author Sundar Subramaniyan
 *
 * @date 9/7/2024
 */

#include <stdio.h>
#include <getopt.h>

/**
 * @brief Default size of the vectors
 */
#define TEST_VECTOR_SIZE (1 << 30)

/**
 * @brief The kernel function that performs the parallel compute operation
 *	  in the GPU
 *
 * @param[in] n Vector size
 * @param[in] a Number to multiply with the vector x
 * @param[in] x Pointer to the device memory holding x vector
 * @param[in, out] y Pointer to the device memory holding y vector
 */
__global__
void saxpy(int n, float a, float *x, float *y)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < n)
		y[i] = a * x[i] + y[i];
}

/**
 * @brief The main function that allocates the vectors in the Host memory
 *	  and initializes them with constant values.
 *
 * Depending on the options passed, the vectors are either copied to the Device memory
 * or mapped from the Host memory and asks the Device to perform parallel computation
 * with the kernel function.
 *
 * @param[in] argc Number of command line arguments passed
 * @param[in] argv The arguments passed to the executable
 *
 * @return 0 for success, -1 for failure
 */
int main(int argc, char *argv[])
{
	int i, ret = 0, N = TEST_VECTOR_SIZE;
	int cuda_memcpy_enabled = true;
	int host_map_enabled = true;
	int c, option_index = 0;
	unsigned int deviceFlags, hostFlags;
	float *x, *y, *d_x, *d_y, *r;
	float maxError = 0.0f;
	hipError_t err;

	/* Parse the command line arguments */
	while (1) {
		struct option long_options[] = {
			/* Options that set a flag */
			{ "no-cuda-memcpy", no_argument, &cuda_memcpy_enabled, 0 },
			{ "no-host-map", no_argument, &host_map_enabled, 0 },

			/* Options that don't set a flag */
			{ "vector-size", required_argument, 0, 's' },
			{ "help", no_argument, 0, 'h' },
		};

		c = getopt_long(argc, argv, "hs:", long_options, &option_index);
		if (c == -1)
			break;

		switch (c) {
		case 0:
			if (long_options[option_index].flag != 0)
				break;
			printf ("option %s", long_options[option_index].name);
			if (optarg)
				printf(" with arg %s", optarg);
			printf("\n");
			break;
		case 's':
			N = atoi(optarg);
			break;
		case 'h':
			/* fallthrough */
		default:
			printf("Usage: %s [--no-cuda-memcpy] [--no-host-map] "\
					"[--vector-size <size>]\n", argv[0]);
			exit(0);
			break;
		}
	}

	/* Get the CUDA Device flags */
	err = hipGetDeviceFlags(&deviceFlags);
	if (err != hipSuccess) {
		printf("Failed to get CUDA device flags: %s/n", hipGetErrorString(err));
		ret = -1;
		goto err_flags;
	}

	/* Interrogate the Device flags and summarize the available support */
	deviceFlags &= cudaDeviceMask;
	printf("CUDA Device Support Summary:\n");
	printf("\tMapped pinned allocations\t\t\t[%s]\n",
			(deviceFlags & hipDeviceMapHost) ? "Yes" : " No");
	printf("\tAutomatic Scheduling\t\t\t\t[%s]\n",
			(deviceFlags & hipDeviceScheduleAuto) ? "Yes" : " No");
	printf("\tUse blocking synchronization\t\t\t[%s]\n",
			(deviceFlags & hipDeviceScheduleBlockingSync) ? "Yes" : " No");
	printf("\tSpin default scheduling\t\t\t\t[%s]\n",
			(deviceFlags & hipDeviceScheduleSpin) ? "Yes" : " No");
	printf("\tYield default scheduling\t\t\t[%s]\n",
			(deviceFlags & hipDeviceScheduleYield) ? "Yes" : " No");
	printf("\tKeep local memory allocation after launch\t[%s]\n",
			(deviceFlags & hipDeviceLmemResizeToMax) ? "Yes" : " No");
	printf("\n");

	/* Show the compute options */
	printf("Compute options:\n");
	printf("\tUsing CUDA Memcpy\t\t\t\t[%s]\n",
			cuda_memcpy_enabled ? "Yes" : " No");
	printf("\tUsing Host Map\t\t\t\t\t[%s]\n",
			((deviceFlags & hipDeviceMapHost)
			 && host_map_enabled) ? "Yes" : " No");
	printf("\tUsing Vector Size\t\t\t\t[%d]\n\n", N);

	/* Setup Host allocation flags */
	hostFlags = hipHostMallocDefault;
	if ((deviceFlags & hipDeviceMapHost) && host_map_enabled) {
		hostFlags |= (hipHostMallocMapped | hipHostMallocWriteCombined);
	}

	/* Allocate x in Host memory */
	err = hipHostAlloc(&x, N * sizeof(float), hostFlags);
	if (err != hipSuccess) {
		printf("Failed to allocate memory for x: %s\n", hipGetErrorString(err));
		ret = -1;
		goto err_alloc_x;
	}

	/* Allocate y in Host memory */
	err = hipHostAlloc(&y, N * sizeof(float), hostFlags);
	if (err != hipSuccess) {
		printf("Failed to allocate memory for y: %s\n", hipGetErrorString(err));
		ret = -1;
		goto err_alloc_y;
	}

	/* Allocate r in Host memory */
	if ((deviceFlags & hipDeviceMapHost) && host_map_enabled)
		hostFlags &= ~hipHostMallocWriteCombined;

	err = hipHostAlloc(&r, N * sizeof(float), hostFlags);
	if (err != hipSuccess) {
		printf("Failed to allocate memory for r: %s\n", hipGetErrorString(err));
		ret = -1;
		goto err_alloc_r;
	}

	/* Allocate x in Device memory */
	err = hipMalloc(&d_x, N * sizeof(float));
	if (err != hipSuccess) {
		printf("Failed to CUDA Malloc for d_x: %s\n", hipGetErrorString(err));
		ret = -1;
		goto err_alloc_d_x;
	}

	/* Allocate y in Device memory */
	err = hipMalloc(&d_y, N * sizeof(float));
	if (err != hipSuccess) {
		printf("Failed to CUDA Malloc for d_y: %s\n", hipGetErrorString(err));
		ret = -1;
		goto err_alloc_d_y;
	}

	/* Initialize the array in Host */
	for (i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	if (cuda_memcpy_enabled) {
		/* Move x from Host to Device memory */
		err = hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
		if (err != hipSuccess) {
			printf("Failed to copy x to d_x: %s\n", hipGetErrorString(err));
			ret = -1;
			goto err_copy;
		}

		/* Move y from Host to Device memory */
		err = hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);
		if (err != hipSuccess) {
			printf("Failed to copy y to d_y: %s\n", hipGetErrorString(err));
			ret = -1;
			goto err_copy;
		}

		/* Perform SAXPY on the vectors copied to the Device memory */
		saxpy<<<(N + 255)/256, 256>>>(N, 2.0f, d_x, d_y);
	} else {
		/* Perform SAXPY on the vectors present in the Host memory */
		saxpy<<<(N + 255)/256, 256>>>(N, 2.0f, x, y);
	}

	/* Wait for the Device to finish */
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		printf("Failed to synchronize: %s\n", hipGetErrorString(err));
		ret = -1;
		goto err_sync;
	}

	if (cuda_memcpy_enabled) {
		/* Copy results from Device to Host memory */
		err = hipMemcpy(r, d_y, N * sizeof(float), hipMemcpyDeviceToHost);
		if (err != hipSuccess) {
			printf("Failed to copy d_y to r: %s\n", hipGetErrorString(err));
			ret = -1;
			goto err_copy;
		}
	} else {
		/* Copy results from Host to Host memory */
		err = hipMemcpy(r, y, N * sizeof(float), hipMemcpyHostToHost);
		if (err != hipSuccess) {
			printf("Failed to copy y to r: %s\n", hipGetErrorString(err));
			ret = -1;
			goto err_copy;
		}
	}

	/* Calculate errors and display the result */
	for (i = 0; i < N; i++)
		maxError = max(maxError, abs(r[i] - 4.0f));

	printf("Compute complete. Max error: %f\n", maxError);

err_sync:
err_copy:
	hipFree(d_y);

err_alloc_d_y:
	hipFree(d_x);

err_alloc_d_x:
	hipHostFree(r);

err_alloc_r:
	hipHostFree(y);

err_alloc_y:
	hipHostFree(x);

err_alloc_x:
err_flags:
	return ret;
}
